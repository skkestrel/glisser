#include "hip/hip_runtime.h"
#include <iomanip>
#include <fstream>
#include <unordered_map>
#include <algorithm>
#include <thread>
#include <unistd.h>
#include <ctime>

#include "swift.h"
#include "util.cuh"
#include "util.h"
#include "types.h"
#include "executor.cuh"
#include "wh.cuh"
#include "convert.h"

namespace sr
{
namespace exec
{
	using namespace sr::wh;
	using namespace sr::util;
	using namespace sr::convert;
	using namespace sr::data;

	// ExecutorData stores partial data that is copied from the GPU - it's used in resync
	ExecutorData::ExecutorData() { }

	ExecutorData::ExecutorData(size_t n)
	{
		r = v = std::vector<f64_3>(n);
		deathflags = std::vector<uint16_t>(n);
		id = std::vector<uint32_t>(n);
		deathtime_index = std::vector<uint32_t>(n);
	}

	struct DeviceParticleUnflaggedPredicate
	{
		template<typename Tuple>
		__host__ __device__
		bool operator()(const Tuple& args)
		{
			uint16_t flag = thrust::get<2>(thrust::get<0>(args));
			return flag == 0;
		}
	};

	struct DeviceParticleAlivePredicate
	{
		template<typename Tuple>
		__host__ __device__
		bool operator()(const Tuple& args)
		{
			uint16_t flag = thrust::get<2>(thrust::get<0>(args));
			return (flag & 0xFE) == 0;
		}
	};

	struct KillEncounterKernel
	{
		KillEncounterKernel() { }

		template<typename Tuple>
		__host__ __device__
		void operator()(Tuple args) const
		{
			uint16_t flags = thrust::get<2>(args);
			if ((flags & 0x01) == 0x01)
			{
				flags = static_cast<uint16_t>((flags & 0xFF00) | 0x80);
			}
			thrust::get<2>(args) = flags;
		}
	};


	Executor::Executor(HostData& _hd, DeviceData& _dd, const Configuration& _config, std::ostream& out)
		: hd(_hd), dd(_dd), output(out), config(_config), resync_counter(0) { }

	// this should be called after hd is populated
	void Executor::init()
	{
		out_timing = std::ofstream(sr::util::joinpath(config.outfolder, "timing.out"));
		out_timing << "t,n_enc,t_swift,t_interp,t_io,t_gpu" << std::endl;

		// glisse only supports helio
		sr::convert::to_helio(hd);

		if (config.interp_planets)
		{
			// setup interpolator
			interpolator = sr::interp::Interpolator(config, hd.planets, config.planet_history_file);

			// step interp forward until we're in the correct interval
			while (interpolator.t1 <= t)
			{
				interpolator.next(hd.planets);
			}

			// make sure we call next at least once, otherwise it means that the lookup file started past
			// the current time

			// interpolator.t0 is initialized to +inf, but calling next once makes it finite
			// so if t0 is not infinity, next was called at least once
			ASSERT(!std::isinf(interpolator.t0), "lookup file starts past the given Initial-Time")

			// we might have started on a point that's in the middle of a lookup interval:
			// the stuff below handles that case

			// cur_ts represents the current timestep number in the current interval
			// even if we start in the middle of the interval, set cur_ts to 0
			interpolator.cur_ts = 0;

			// n_ts is the total number of timesteps in the current interval
			// the number of timesteps is the remaining time in the interval divided by dt
			interpolator.n_ts = std::max<size_t>(1, static_cast<size_t>(std::round((interpolator.t1 - t) / config.dt)));

			// set the effective dt appropriately
			interpolator.eff_dt = (interpolator.t1 - t) / static_cast<double>(interpolator.n_ts);

			// relative time since the beginning of the interval
			interpolator.rel_t = t - interpolator.t0;

			// fill the planet location at t=0
			interpolator.fill_one(hd.planets, interpolator.rel_t);
		}

		// setup integrator - this gives all particles an acceleration and also detects initial encounters
		// needs to happen after interpolator init to get initial planet positions
		integrator = sr::wh::WHCudaIntegrator(hd.planets, hd.particles, config, htd_stream);

		// setup encounter integrator
		swift = sr::swift::SwiftEncounterIntegrator(config, hd.particles.n());


		// calculate initial energy
		calculate_planet_metrics(hd.planets, &e_0, nullptr);

		output << std::setprecision(7);
		output << "e_0 (planets) = " << e_0 << std::endl;
		output << "n_particle = " << hd.particles.n() << std::endl;
		output << "n_particle_alive = " << hd.particles.n_alive() << std::endl;
		output << "==================================" << std::endl;
		output << "Sending initial conditions to GPU." << std::endl;

		// create cuda streams
		hipStreamCreate(&main_stream);
		hipStreamCreate(&htd_stream);
		hipStreamCreate(&dth_stream);

		hipEventCreate(&start_event);
		hipEventCreate(&gpu_finish_event);

		// more initialization
		rollback_state = DeviceParticlePhaseSpace(hd.particles.n());
		dd.particles = DeviceParticlePhaseSpace(hd.particles.n());

		// having two planet data arrays allows us to update planet data while the other one is
		// being used by the integrator, so there is always one being used to integrate while
		// the other one is used to upload data for the next timechunk
		dd.planets0 = DevicePlanetPhaseSpace(hd.planets.n(), config.tbsize);
		dd.planets1 = DevicePlanetPhaseSpace(hd.planets.n(), config.tbsize);

		// planet_data_id chooses whether to use planets0 or planets1, arbitrarily choose 0 to start with
		dd.planet_data_id = 0;

		// ** INITIALIZE PARTICLES


		// partition alive
		hd.particles.stable_partition_unflagged(0, hd.particles.n());

		// now, n_alive = n_unflagged
		size_t n_alive = hd.particles.n_alive();

		// bring the encounter particles to the beginning
		hd.particles.stable_partition_alive(n_alive, hd.particles.n() - n_alive);
		// now, n_alive = total alive (enc + nonenc)

		// don't need to upload if no particles
		if (hd.particles.n() > 0)
		{
			upload_data(0, hd.particles.n());
			dd.particle_phase_space().n_alive = hd.particles.n_alive();
		}

		// download data right after uploading -
		// i forgot why we need this, but I think it's just so the GPU data can be debugged
		download_data();

		starttime = std::chrono::high_resolution_clock::now();

		output << "       Starting simulation.       " << std::endl << std::endl;

		if (encounter_output)
		{
			*encounter_output << std::setprecision(17);
		}

		hd.planets_snapshot = hd.planets.base;

		// upload planet data before the first timechunk
		update_planets();
	}

	void Executor::swap_logs()
	{
		// swap new and old logs
		hd.planets.swap_logs();
		integrator.swap_logs();
	}

	void Executor::update_planets()
	{
		prev_dt = cur_dt;
		prev_tbsize = cur_tbsize;

		integrator.recalculate_rh(hd.planets);

		if (config.interp_planets)
		{
			// cur_ts = n_ts means that we've reached the end of the lookup interval
			// need to step to the next one
			if (interpolator.cur_ts == interpolator.n_ts)
			{
				// sanity check - make sure that rel_t = t1 - t0
				double diff = interpolator.rel_t - (interpolator.t1 - interpolator.t0);
				ASSERT(std::abs(diff) < 1e-8, "sanity check failed - adjusting t by too much: " + std::to_string(diff) +)

				// step
				interpolator.next(hd.planets);

				// reset cur_ts
				interpolator.cur_ts = 0;

				// force the integration time to be equal to the interval begin time
				t = interpolator.t0;

				// now we mark that we just stepped forward in the interpolator - resync uses this
				// to figure out whether to do the "ending encounter resolution" on swift
				starting_lookup_interval = true;
			}

			// this is for the singular case at the very start of the integration, cur_ts is set to 0
			else if (interpolator.cur_ts == 0)
			{
				starting_lookup_interval = true;
			}
			else
			{
				starting_lookup_interval = false;
			}

			// select dt
			cur_dt = interpolator.eff_dt;

			// select the size of the next timestep
			// it can be no more than tbsize, and cannot go past the end of the interval
			cur_tbsize = std::min(config.tbsize, static_cast<uint32_t>(interpolator.n_ts - interpolator.cur_ts));

			// fill the planet logs

			interpolator.fill(hd.planets, cur_tbsize, interpolator.rel_t, cur_dt);

			// advance rel_t
			interpolator.rel_t += static_cast<double>(cur_tbsize) * cur_dt;

			// advance cur_ts
			interpolator.cur_ts += cur_tbsize;

			// since we interpolated planet positions, the planet accelerations were not calculated
			// planet accelerations are needed to calculate h0 which is a term used on the GPU
			// load h0 manually - this doesn't need to happen if using the normal planet integrator
			integrator.load_h0(hd.planets);

			// make sure that we haven't overrun the lookup interval
			ASSERT(interpolator.cur_ts <= interpolator.n_ts, "sanity check fialed - interpolator cur_ts is in an illegal position")
		}
		else
		{
			cur_dt = config.dt;
			cur_tbsize = config.tbsize;

			integrator.integrate_planets_timeblock(hd.planets, cur_tbsize, t, cur_dt);
		}

		// swap new and old logs:
		// the interpolator and integrator both make sure to write to the old logs
		// so swapping the logs here brings the logs into the correct position
		// of course, the GPU integration next chunk is done using the new logs
		swap_logs();

		// we only upload the planet log if any particles are going to use the planet log on the GPU
		// i.e. there are particles that could be alive
		if (dd.particle_phase_space().n_alive > 0 || hd.particles.n_encounter() > 0)
		{
			upload_planet_log();
		}
	}

	void Executor::upload_data(size_t begin, size_t length)
	{
		auto& particles = dd.particle_phase_space();
		integrator.upload_data_cuda(htd_stream, begin, length);

		memcpy_htd(particles.r, hd.particles.r(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.v, hd.particles.v(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.deathflags, hd.particles.deathflags(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.deathtime_index, hd.particles.deathtime_index(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.id, hd.particles.id(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
	}

	void Executor::add_job(const std::function<void()>& job)
	{
		work.push_back(std::move(job));
	}

	void Executor::download_data()
	{
		auto& particles = dd.particle_phase_space();

		// download only the alive particle data - dead particles are handled in resync()
		// since they're dead, they don't get updated any more so no need to download again

		// note: dead particles DO need to be downloaded when using resync2 so we might as well just download everything
		memcpy_dth(hd.particles.r(), particles.r, dth_stream, 0, 0, particles.n_total);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.v(), particles.v, dth_stream, 0, 0, particles.n_total);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.id(), particles.id, dth_stream, 0, 0, particles.n_total);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.deathflags(), particles.deathflags, dth_stream, 0, 0, particles.n_total);
		hipStreamSynchronize(dth_stream);

		// host n_alive includes encounter particles, but not the device n_alive
		hd.particles.n_alive() = dd.particle_phase_space().n_alive + hd.particles.n_encounter();
	}

	void Executor::upload_planet_log()
	{
		// alternate the planet data id, this is to make sure we don't copy into data currently being used on GPU
		dd.planet_data_id++;

		// planet_phase_space uses planet_data_id to figure out which one to get
		auto& planets = dd.planet_phase_space();

		// copy in everything
		memcpy_htd(planets.r_log, hd.planets.r_log().log, htd_stream);
		memcpy_htd(planets.m, hd.planets.m(), htd_stream);
		memcpy_htd(planets.id, hd.planets.id(), htd_stream);

		hipStreamSynchronize(htd_stream);

		planets.n_alive = hd.planets.n_alive();
		planets.log_len = hd.planets.r_log().len;

		integrator.upload_planet_log_cuda(htd_stream, dd.planet_data_id);
	}

	double Executor::time() const
	{
		auto now = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> millis = now - starttime;
		return millis.count() / 60000;
	}

	void Executor::handle_encounters(bool called_from_resync)
	{
		size_t encounter_start = hd.particles.n_alive() - hd.particles.n_encounter();

		// if at the beginning of a lookup interval, don't integrate the previous chunk
		// IGNORE IF CALLED_FROM_RESYNC
		// prev_len = 0 means the previous chunk isn't integrated
		size_t prev_len = prev_tbsize;
		if (starting_lookup_interval && !called_from_resync)
		{
			prev_len = 0;
		}

		// if called from resync, don't integrate the future chunk - this is because
		// the future chunk is in a different lookup interval
		size_t cur_len = called_from_resync ? 0 : cur_tbsize;

		// if called from resync, update_planets will have updated everything to the next chunk
		// already, so we need to use prev_dt
		double which_dt = called_from_resync ? prev_dt : cur_dt;

		if (called_from_resync)
		{
			ASSERT(std::abs(interpolator.t0 - t) < 1e-2, "sanity check failed: end-of-chunk encounter time")
		}

		// if called from resync, t is at the end of the timechunk, otherwise use interpolator relative t MINUS a timeblock because rel_t is the planet time, not the particle time
		double rel_t = called_from_resync ? interpolator.t0 - interpolator.t_m1 : interpolator.rel_t - interpolator.eff_dt * static_cast<double>(cur_tbsize);

		double which_t = called_from_resync ? interpolator.t0 : interpolator.t0 + rel_t;


		swift.begin_integrate(hd.planets, hd.particles, interpolator, called_from_resync, which_t, rel_t, which_dt, prev_len, cur_len);
		
		// if this was called in the middle of loop, we do the work here while other processes are happening
		// ** temporarily disabled this parallielization for profiling
/*
		if (!called_from_resync)
		{
			for (auto& i : work) i();
			work.clear();
		}
*/

		// update encounter particles
		swift.end_integrate(hd.particles);

		ASSERT(std::isnormal(hd.particles.r()[0].x), "nan")

		// whether to use the old log or not depends on whether we called from resync
		size_t which_timestep_index = (called_from_resync ? hd.planets.r_log().len_old : hd.planets.r_log().len) - 1;

		// need to calculate particle accelerations for the next timeblock -
		// this is because these particles did not come out of a regular GPU timechunk,
		// so accelerations are outdated

		// load accelerations (the planets already have h0 loaded, so no problem here)
		integrator.helio_acc_particles(
			hd.planets,
			hd.particles,
			encounter_start,
			hd.particles.n_encounter(),
			t + static_cast<double>(cur_len) * which_dt,
			which_timestep_index,
			called_from_resync // use the old log if called from resync, otherwise use the new log
		);

		// since helio_acc_particles sets deathflags, unset them IFF in encounter since we want the GPU to detect an encounter, delayed
		// however, if in resync, we wan to do to the next one immediately to start the next history interval
		// if (!called_from_resync)
		{
			for (size_t i = encounter_start; i < hd.particles.n_encounter(); i++)
			{
				if ((hd.particles.deathflags()[i] & 0xFF) == 0x01)
				{
					hd.particles.deathflags()[i] = 0;
				}
			}
		}

		// upload the changes to the GPU
		// no need to sort the particles here, resync will do all the sorting
		upload_data(encounter_start, hd.particles.n_encounter());

		// set n_alive so that the resync function knows to deal with the particles that we just added back
		dd.particles.n_alive = hd.particles.n_alive();

		download_data();
	}

	bool Executor::loop(double* cputimeout, double* gputimeout)
	{
		// At the beginning of the loop the following things should be true:
		// No GPU or CPU processes are running
		// The next timeblock should be loaded on the GPU
		// cur_tbsize refers to the size of that timeblock
		// prev_tbsize refers to the size of the previous timeblock
		// planet logs (old and new are filled with prev_tbsize and cur_tbsize entries, respectively)
		// particles on the GPU have an acceleration loaded (integerator.particle_a)
		// -> this can come from either the previous GPU kernel run, or, the WHIntegrator constructor
		// the time refers to the time before the next GPU kernel, which is also equal to the time at the start of the current planet log
		// it's possible that not all particles have reached the current t, since they might be about to be stepped forward on SWIFT all the way until
		// t + cur_tbsize * dt
		// t = the time at the start of the block that is about to be calculated

		std::clock_t c_start = std::clock();

		if (dd.particle_phase_space().n_alive > 0)
		{
			// if resolving encounters, we need the particle states at the beginning of the chunk
			// so that encounter particles can be rolled back to their initial state
			if (config.resolve_encounters)
			{
				memcpy_dtd(rollback_state.r, dd.particle_phase_space().r, main_stream);
				memcpy_dtd(rollback_state.v, dd.particle_phase_space().v, main_stream);
				memcpy_dtd(rollback_state.deathflags, dd.particle_phase_space().deathflags, main_stream);
				memcpy_dtd(rollback_state.deathtime_index, dd.particle_phase_space().deathtime_index, main_stream);
				memcpy_dtd(rollback_state.id, dd.particle_phase_space().id, main_stream);

				rollback_state.n_alive = dd.particle_phase_space().n_alive;
				rollback_state.n_total = dd.particle_phase_space().n_total;
			}

			hipEventRecord(start_event, main_stream);

			// in order to integrate the particles on GPU, the particle accelerations must be set.
			// typically the accelerations are set by the previous timeblock
			// but in the case of the first timeblock, or when recovering from a close encounter, it needs to be set manually...

			integrator.integrate_particles_timeblock_cuda(
				main_stream,
				dd.planet_data_id,
				dd.planet_phase_space(),
				dd.particle_phase_space(),
				cur_dt
			);

			hipEventRecord(gpu_finish_event, main_stream);
		}

		for (auto& i : work) i();
		work.clear();

		size_t n_encounter_start = hd.particles.n_encounter();

		float worktime = static_cast<float>(std::clock() - c_start) / CLOCKS_PER_SEC * 1000;

		// do work after GPU starts
		// this is typically all file I/O
		// when encounters are enabled, handle_encounters handles the work vector
		if (config.resolve_encounters && hd.particles.n_encounter() > 0)
		{
			handle_encounters(false);
		}
		else
		{
			/*
			for (auto& i : work) i();
			work.clear();
			*/
		}

		float encountertime = (static_cast<float>(std::clock() - c_start) / CLOCKS_PER_SEC * 1000) - worktime;

		// The snapshot contains the planet states at the end of the current timechunk (= beginning of next timechunk)
		// this is necessary since update_planets brings all the planets one timechunk forward
		// e.g. if the integration finishes at t=1, update_planets will still bring the planets forward to t=1 + dt
		// so in order to get the correct planetary positions, we need to record the planet positions before they get updated
		hd.planets_snapshot = hd.planets.base;

		// step time forward
		t += cur_dt * static_cast<double>(cur_tbsize);

		// calculate planetary positions for the next chunk - this is REALLY important and also very subtle
		// usually, this would be called at the very end of loop() but
		// we can save some time by doing this here
		// HOWEVER, doing this here means that the planets are now one entire time chunk ahead of the particles
		// for the remainder of loop()
		// it's VERY important to make sure that we're using the correct data

		// IF A FUNCTION IS CALLED AFTER UPDATE_PLANETS: USE OLD DATA (e.g. prev_dt, prev_tbsize, pl.r_log.get<old=true>, etc...)
		update_planets();

		float updatetime = (static_cast<float>(std::clock() - c_start) / CLOCKS_PER_SEC * 1000) - encountertime - worktime;

		if (cputimeout) *cputimeout = static_cast<float>(std::clock() - c_start) / CLOCKS_PER_SEC * 1000;

		float gputime = 0;

		// there's nothing to resync if the GPU didn't integrate any particles, i.e. dd.particles.n_alive = 0
		if (dd.particle_phase_space().n_alive > 0)
		{
			hipStreamSynchronize(main_stream);
			hipStreamSynchronize(htd_stream);
			hipEventSynchronize(gpu_finish_event);

			download_data();

			hipEventElapsedTime(&gputime, start_event, gpu_finish_event);
			if (gputimeout) *gputimeout = gputime;

			// resync_every is guaranteed to be 1 if encounters are enabled
			resync_counter++;
			if (resync_counter % config.resync_every == 0)
			{
				resync2();
			}
		}

		out_timing << t << " " << n_encounter_start << " " << encountertime << " " << updatetime << " " << worktime << " " << gputime << std::endl;

		// if not resolving encounters, every time is safe to end on
		// if resolving encounters, only timechunks that end the lookup interval are safe
		return starting_lookup_interval || !config.resolve_encounters;
	}

	void Executor::resync2()
	{
		// this is a simplified version of the resync algorithm which reads the entire particle arrays, this
		// means that the algorithm doesn't need to match particle ids when reading

		auto& particles = dd.particle_phase_space();
		size_t prev_alive = particles.n_alive;

		// kill particles in encounters
		if (!config.resolve_encounters)
		{
			auto it = particles.begin();
			thrust::for_each(thrust::cuda::par.on(main_stream), it, it + particles.n_alive, KillEncounterKernel());
		}

		// partition twice
		if (config.resolve_encounters)
		{
			auto partition_it = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), rollback_state.begin(), integrator.device_begin()));

			particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(main_stream),
					partition_it, partition_it + particles.n_alive, DeviceParticleUnflaggedPredicate()) - partition_it;
			
			// the second partition for encounter particles only needs to run between n_alive and prev_alive, since all the alive particles
			// will be pushed to the beginning anyways
			hd.particles.n_encounter() = (thrust::stable_partition(thrust::cuda::par.on(main_stream),
					partition_it + particles.n_alive, partition_it + prev_alive, DeviceParticleAlivePredicate()) - partition_it) - particles.n_alive;

			hipStreamSynchronize(main_stream);
		}
		else
		{
			auto partition_it = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), integrator.device_begin()));
			particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(main_stream),
					partition_it, partition_it + particles.n_alive, DeviceParticleUnflaggedPredicate()) - partition_it;

			hipStreamSynchronize(main_stream);
		}

		// copy everything back - n_alive is also copied from device to host
		download_data();

		// set the deathtime for dead particles - let's set the encounter particles deathtimes too, just to show when they entered encounter
		// here t refers to the ending time of the timechunk
		for (size_t i = particles.n_alive; i < prev_alive; i++)
		{
			// t = time at the end of this chunk
			hd.particles.deathtime_map()[hd.particles.id()[i]] = static_cast<float>(t);

			if (hd.particles.deathflags()[i] & 0x04) 
			{
				output << "warning - particle " << hd.particles.id()[i] << " did not converge on GPU" << std::endl;
			}

			if (encounter_output)
			{
				if (hd.particles.deathflags()[i] & 0x80)
				{
					*encounter_output << hd.particles.id()[i] << " death " << t << std::endl;
				}
				else
				{
					*encounter_output << hd.particles.id()[i] << " encounter " << t << std::endl;
				}
			}
		}

		// for encounter particles, use the rollback data
		if (hd.particles.n_encounter() > 0)
		{
			memcpy_dth(hd.particles.r(), rollback_state.r, dth_stream, particles.n_alive, particles.n_alive, hd.particles.n_encounter());
			hipStreamSynchronize(dth_stream);
			memcpy_dth(hd.particles.v(), rollback_state.v, dth_stream, particles.n_alive, particles.n_alive, hd.particles.n_encounter());
			hipStreamSynchronize(dth_stream);
		}

		// handle particles that just entered encounter, and partition again
		if (starting_lookup_interval && hd.particles.n_encounter() > 0)
		{
			hipDeviceSynchronize();

			handle_encounters(true);

			auto partition_it = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), rollback_state.begin(), integrator.device_begin()));

			particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(main_stream),
					partition_it, partition_it + particles.n_alive, DeviceParticleUnflaggedPredicate()) - partition_it;
			
			hd.particles.n_encounter() = (thrust::stable_partition(thrust::cuda::par.on(main_stream),
					partition_it + particles.n_alive, partition_it + prev_alive, DeviceParticleAlivePredicate()) - partition_it) - particles.n_alive;

			hipStreamSynchronize(main_stream);

			download_data();
		}
	}


	void Executor::finish()
	{
		hipStreamSynchronize(main_stream);
		swift.write_stat(sr::util::joinpath(config.outfolder, "stat.out"));


		for (auto& i : work) i();
		work.clear();

		output << "Simulation finished. t = " << t << ". n_particle = " << hd.particles.n_alive() << std::endl;
	}
}
}
